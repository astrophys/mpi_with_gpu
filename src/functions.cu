/**************************************************************
 *  AUTHOR : Ali Snedden
 *  DATE   : 02/21/24
 *  LICENSE: GPL-3
 *  PURPOSE: 
 *      This is a file with helper functions used by both cpu
 *       and gpu versions of this code
**************************************************************/
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
using namespace std;



/********************************************************
    ARGS:
        message : char array
    DESCRIPTION:
        Print out error, exit with error
    RETURN:
    DEBUG:
    NOTES: 
    FUTURE:
*******************************************************/
void exit_with_error(char * message){
    fprintf(stderr, "%s", message);
    fflush(stderr);
    exit(1);
}


/**********************************
ARGS:
    array1D : 'flattened' 2D array as 1D
    N       : length of array
RETURN:
    N/A
DESCRIPTION:
    Prints 1D array and 3D coords
DEBUG:
    1. spot checked, it works
FUTURE:
***********************************/
void write_1D_array(float * array1D, int Nx, int Ny, FILE * f){
    int i = 0;
    int j = 0;
    int idx = 0;
    for(i=0; i<Nx; i++){
        for(j=0; j<Ny; j++){
            idx = map_idx(i,j,Ny);
            fprintf(f, "%*.1f ", 5, array1D[idx]);
        }
        fprintf(f, "\n");
    }
}


/**********************************
ARGS:
    array1D : 'flattened' 2D array as 1D
    N       : length of array
RETURN:
    N/A
DESCRIPTION:
    Prints 1D array and 3D coords
DEBUG:
    1. spot checked, it works
FUTURE:
***********************************/
void print_1D_array(float * array1D, int Nx, int Ny){
    int i = 0;
    int j = 0;
    int idx = 0;
    for(i=0; i<Nx; i++){
        for(j=0; j<Ny; j++){
            idx = map_idx(i,j,Ny);
            printf("%*.1f ", 5, array1D[idx]);
        }
        printf("\n");
    }
}


/********************************************************
    ARGS:
    DESCRIPTION:
    RETURN:
    DEBUG:
    NOTES: 
        1. Use 'flattened' 2D array
    FUTURE:
*******************************************************/
void initialize_matrix(float *A, int * dim, float value){
    for(int i=0; i<dim[0]; i++){
        for(int j=0; j<dim[1]; j++){
            //A[i*dim[0]+j] = value;
            A[map_idx(i,j,dim[1])] = value;
        }       
    }

}


// This is C++ code - from stackoverflow : https://stackoverflow.com/q/14038589 
/********************************************************
    ARGS:
        hipError_t code
        const char* file : 
        int line :
    DESCRIPTION:
        Uses macro and inline function b/c it is important to preserve the
        file and line number in the error printing.
    RETURN:
    DEBUG:
    NOTES: 
    FUTURE:
*******************************************************/
#define gpuErrChk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s : %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}




