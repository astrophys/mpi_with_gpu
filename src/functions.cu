#include "hip/hip_runtime.h"
/**************************************************************
 *  AUTHOR : Ali Snedden
 *  DATE   : 02/21/24
 *  LICENSE: GPL-3
 *  PURPOSE: 
 *      This is a file with helper functions used by both cpu
 *       and gpu versions of this code
**************************************************************/
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <string.h>
#include <mpi.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
//using namespace nvcuda; 
using namespace std; 



/********************************************************
    ARGS:
        message : char array
    DESCRIPTION:
        Print out error, exit with error
    RETURN:
    DEBUG:
    NOTES: 
    FUTURE:
*******************************************************/
void exit_with_error(char * message){
    fprintf(stderr, "%s", message);
    fflush(stderr);
    MPI_Finalize();
    exit(1);
}


/**********************************
ARGS:
RETURN:
DESCRIPTION:
    Map 2D indices to 1D index
DEBUG:
    1. read_numpy_matrix() uses this function extensively.
       Directly compared output from read_numpy_matrix() with input
       and was IDENTICAL. This could not work if map_idx() didn't 
       function correctly.
FUTURE:
    1. Add error checking if not too expensive
***********************************/
int map_idx(int i, int j, int Ny){
    return (Ny * i + j);
}
// Visible to device
__device__ int d_map_idx(int i, int j, int Ny){
    return (Ny * i + j);
}


/**********************************
ARGS:
    array1D : 'flattened' 2D array as 1D
    N       : length of array
RETURN:
    N/A
DESCRIPTION:
    Prints 1D array and 3D coords
DEBUG:
    1. spot checked, it works
FUTURE:
***********************************/
void write_1D_array(float * array1D, int Nx, int Ny, FILE * f){
    int i = 0;
    int j = 0;
    int idx = 0;
    for(i=0; i<Nx; i++){
        for(j=0; j<Ny; j++){
            idx = map_idx(i,j,Ny);
            fprintf(f, "%*.1f ", 5, array1D[idx]);
        }
        fprintf(f, "\n");
    }
}


/**********************************
ARGS:
    array1D : 'flattened' 2D array as 1D
    N       : length of array
RETURN:
    N/A
DESCRIPTION:
    Prints 1D array and 1D coords
DEBUG:
    1. spot checked, it works
FUTURE:
***********************************/
void print_1D_array(float * array1D, int Nx, int Ny){
    int i = 0;
    int j = 0;
    int idx = 0;
    for(i=0; i<Nx; i++){
        for(j=0; j<Ny; j++){
            idx = map_idx(i,j,Ny);
            printf("%*.1f ", 5, array1D[idx]);
        }
        printf("\n");
    }
}


/********************************************************
    ARGS:
    DESCRIPTION:
    RETURN:
    DEBUG:
    NOTES: 
        1. Use 'flattened' 2D array
    FUTURE:
*******************************************************/
void initialize_matrix(float *A, int * dim, float value){
    for(int i=0; i<dim[0]; i++){
        for(int j=0; j<dim[1]; j++){
            //A[i*dim[0]+j] = value;
            A[map_idx(i,j,dim[1])] = value;
        }       
    }
}


/********************************************************
    ARGS:
        float * A    : 1D projection of 2D matrix
        int * dim    : x and y dimensions
        float factor : factor to multiply identity matrix by
    DESCRIPTION:
        Initialize identity matrix
    RETURN:
    DEBUG:
    NOTES: 
        1. Use 'flattened' 2D array
    FUTURE:
*******************************************************/
void identity_matrix(float *A, int * dim, float factor){
    for(int i=0; i<dim[0]; i++){
        for(int j=0; j<dim[1]; j++){
            //A[i*dim[0]+j] = value;
            if(i == j){
                A[map_idx(i,j,dim[1])] = 1.0 * factor;
            }else{
                A[map_idx(i,j,dim[1])] = 0;
            }
        }       
    }
}


/**********************************
ARGS:
    int * A : flattened 2D array
    int M   : number of Rows
    int N   : number of Cols
RETURN:
DESCRIPTION:
    Print 2D matrix. Must do it to on device b/c the halfs must
    be converted to ints __and__ that can __only__ be done on 
    the device. It is ridiculous, but I'm only using __one__ 
    thread to print the matrix.
DEBUG:
FUTURE:
***********************************
__global__ void print_matrix(half * A, int M, int N){
    int i = 0;
    int j = 0;
    int rIdx = blockIdx.x * blockDim.x + threadIdx.x;     //Row    index
    int cIdx = blockIdx.y * blockDim.y + threadIdx.y;     //Column index
    
    if(rIdx == 0 && cIdx == 0){
        for(i=0; i<M; i++){
            for(j=0; j<N; j++){
                printf("%*i", 3, __half2int_rd(A[d_map_idx(i,j,N)]));
            }
            printf("\n");
        }
    }
}*/


/**********************************
ARGS:
    int * A : flattened 2D array - Input array to convert
    half * B: flattened 2D array - Result
    int M   : number of Rows
    int N   : number of Cols
RETURN:
DESCRIPTION:
    Print 2D matrix
DEBUG:
FUTURE:
***********************************
__global__ void some_func(half * A){
    int startIdx = blockIdx.x * blockDim.x + threadIdx.x; // Index of current thread in block
    int stride   = blockDim.x * gridDim.x;                // Number of threads in the block
    //printf("%i : %i : %i \n", startIdx, stride, threadIdx.x);

    if(blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x ==1 && threadIdx.y == 1){
        printf("****************************\n\tblockDim.x = %i\n\tblockDim.y = %i\n\tgridDim.x = %i\n\tgridDim.y = %i\n\tblockIdx.x = %i\n\tblockIdx.y = %i\n\tthreadIdx.x = %i\n\tthreadIdx.y = %i\n",
               blockDim.x, blockDim.y, gridDim.x, gridDim.y, blockIdx.x, blockIdx.y,
               threadIdx.x, threadIdx.y);
    }
}*/



// This is C++ code - from stackoverflow : https://stackoverflow.com/q/14038589 
/********************************************************
    ARGS:
        hipError_t code
        const char* file : 
        int line :
    DESCRIPTION:
        Uses macro and inline function b/c it is important to preserve the
        file and line number in the error printing.
    RETURN:
    DEBUG:
    NOTES: 
    FUTURE:
*******************************************************/
#define gpuErrChk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s : %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


/**********************************
ARGS:
    int exitval : exit value
RETURN:
DESCRIPTION:
    Parses Command line options. Prints 'help' if requested.
DEBUG:
FUTURE:
    1. Learn an argparse like lib
***********************************/
void print_help(int exitval){
    printf("USAGE:\n\n");
    printf("mpiexec --np num mpi_matrix_mult --option option\n");
    printf("\tnum    = (int) number of MPI tasks\n");
    printf("\toption = (str) 'mpi_cpu', 'mpi_cache_opt', 'mpi_openmp_cpu' or\n");
    printf("\t         'mpi_gpu' or 'mpi_openmp_cpu_opt'\n");
    exit(exitval);
}


/**********************************
ARGS:
    char * argv[] : CL args to parse 
RETURN:
DESCRIPTION:
    Parses Command line options. Prints 'help' if requested.
DEBUG:
FUTURE:
    1. Learn an argparse like lib
***********************************/
char * parse_cl_options(char ** argv){
    /*************************** Help Section ***************************/
    if(argv[1][1] == 'h' && argv[1][0] == '-'){
        print_help(0);
    }
    char * option = NULL;

    // There must be a better way in C++
    if(strcmp("--option", argv[1]) == 0){
        printf("%s\n", argv[2]);
        option = argv[2];
        fflush(stdout);
    }else{
        print_help(1);
    }
    
    return(option);
}
